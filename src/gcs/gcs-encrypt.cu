#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <stdlib.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include <vector>
#include <ctime>

#include "../include/aeslib.hpp"
#include "../include/genlib.hpp"
#include "../include/parallelcore.cuh"

using namespace std;

void GCNS(vector<byte *> &uData, vector<int> &uLens, vector<byte *> &uKeys, vector<byte *> &ciphers) {
    
    // The published algorithm copies the ciphers back to uData
    // But I'm gonna put them in a separate array in case I need the raw user data for something.

    // The following variables are stored in global memory
    // They will be further copied to shared memory in the kernel
    // The idea being to reduce memory latency 
    byte *d_sbox;
    byte *d_mul2;
    byte *d_mul3;
    byte *d_rcon;

    CUDA_ERR_CHK(hipMalloc((void **) &d_sbox, 256));
    CUDA_ERR_CHK(hipMalloc((void **) &d_mul2, 256));
    CUDA_ERR_CHK(hipMalloc((void **) &d_mul3, 256));
    CUDA_ERR_CHK(hipMalloc((void **) &d_rcon, 256));

    CUDA_ERR_CHK(hipMemcpy(d_sbox, sbox, 256, hipMemcpyHostToDevice));
    CUDA_ERR_CHK(hipMemcpy(d_mul2, mul2, 256, hipMemcpyHostToDevice));
    CUDA_ERR_CHK(hipMemcpy(d_mul3, mul3, 256, hipMemcpyHostToDevice));
    CUDA_ERR_CHK(hipMemcpy(d_rcon, rcon, 256, hipMemcpyHostToDevice));

    int n = uData.size();
    byte *h_uData[n];
    byte *h_uKeys[n];

    for(int i = 0; i < n; i++) {
        CUDA_ERR_CHK(hipMalloc((void **) &h_uData[i], uLens[i]));
        CUDA_ERR_CHK(hipMalloc((void **) &h_uKeys[i], 16));
        CUDA_ERR_CHK(hipMemcpy(h_uData[i], uData[i], uLens[i], hipMemcpyHostToDevice))
        CUDA_ERR_CHK(hipMemcpy(h_uKeys[i], uKeys[i], 16, hipMemcpyHostToDevice));
    }

    byte **d_uData;
    byte **d_uKeys;
    int *d_uLens;
    CUDA_ERR_CHK(hipMalloc((void **) &d_uData, n*sizeof(byte*)));
    CUDA_ERR_CHK(hipMalloc((void **) &d_uKeys, n*sizeof(byte*)));
    CUDA_ERR_CHK(hipMalloc((void **) &d_uLens, n*sizeof(int)));
    CUDA_ERR_CHK(hipMemcpy(d_uData, h_uData, n, hipMemcpyHostToDevice));
    CUDA_ERR_CHK(hipMemcpy(d_uKeys, h_uKeys, n, hipMemcpyHostToDevice));
    CUDA_ERR_CHK(hipMemcpy(d_uLens, &(uLens[0]), n, hipMemcpyHostToDevice));

    
    int gridsize, blocksize;
    blocksize = BLOCKSIZE;
    gridsize = n; 
    GCS_Cipher <<< gridsize, blocksize>>> (d_uData, d_uKeys, d_uLens, n, d_sbox, d_mul2, d_mul3, d_rcon);
    CUDA_ERR_CHK(hipPeekAtLastError());
    
    for(int i = 0; i < n; i++) {
        byte *cipher = new byte[uLens[i]];
        CUDA_ERR_CHK(hipMemcpy(cipher, h_uData[i], uLens[i], hipMemcpyDeviceToHost));
        ciphers.push_back(move(cipher));
        CUDA_ERR_CHK(hipFree(h_uData[i]));
        CUDA_ERR_CHK(hipFree(h_uKeys[i]));
    }

    CUDA_ERR_CHK(hipFree(d_uData));
    CUDA_ERR_CHK(hipFree(d_uKeys));
    CUDA_ERR_CHK(hipFree(d_uLens));
}

void get_data(opts vars, vector<byte*> &msgs, vector<int> &lens, vector<byte*> &keys, int i, int j) {

    if(i < vars.n_files_start || i > vars.n_files_end || j < 0 || j >= vars.m_batches ) {
        cout << "Invalid getdata params";
        return;
    }

	string msg_path, key_path;
    ifstream f_msg, f_key;

    int k, n;
    for(k = 0; k < i; k++) {
        msg_path = vars.path + "/" + to_string(i) + "/" + to_string(j) + "/" + to_string(k);
        key_path = msg_path+"_key";

        f_msg.open(msg_path, ios::binary);
        f_key.open(key_path, ios::binary);

	    if(f_msg && f_key) {

		    f_msg.seekg(0, f_msg.end);
	        n = f_msg.tellg();
    		f_msg.seekg(0, f_msg.beg);

            byte *message = new byte[n];
		    byte *key = new byte[16];

            f_msg.read( reinterpret_cast<char *> (message), n);
		    f_key.read( reinterpret_cast<char *> (key), 16);

            msgs.push_back(move(message));
            lens.push_back(n);
            keys.push_back(move(key));

            f_msg.close();
            f_key.close();
        }
        else {
            cout << "read failed";
        }
    }
}


int main() {
    opts vars = get_defaults();
	clock_t start, end;
    int i, j;
    for(i = vars.n_files_start; i <= vars.n_files_end; i += vars.step) {
        
        long long isum = 0;
        for(j = 0; j < vars.m_batches; j++) {
            vector<long> batchtimes;
			long sum = 0;
            
            vector<byte*> uData;
            vector<int> uLens;
            vector<byte*> uKeys;

            get_data(vars, uData, uLens, uKeys, i, j);
            vector<byte*> ciphers;
            ciphers.reserve(i);
            
            start = clock();
            GCNS(uData, uLens, uKeys, ciphers);
            end = clock();
            batchtimes.push_back((end-start));
			sum += (end-start);
			printf("\n N_FILES: %5d | BATCH: %2d | TIME: %10.4lf ms", i, j, ((double)sum * 100)/CLOCKS_PER_SEC);
			isum += sum;

            string out_path;
            ofstream fout;
            for(int k = 0; k < i; k++) {
                out_path = vars.path + "/" + to_string(i) + "/" + to_string(j) + "/" + to_string(k) + "_cipher_gcs";
                fout.open(out_path, ios::binary);
                fout.write(reinterpret_cast<char *> (ciphers[k]), uLens[k]);
                fout.close();
                delete[] uData[k];
                delete[] uKeys[k];
            }
        }
		printf("\n N_FILES: %5d | AVG_TIME: %10.4lf ms\n", i, (((double)isum * 100)/vars.m_batches)/CLOCKS_PER_SEC);
    }

    return 0;
}