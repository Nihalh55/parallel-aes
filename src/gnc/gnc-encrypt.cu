#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <stdlib.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include <vector>
#include <ctime>

#include "../include/aeslib.hpp"
#include "../include/genlib.hpp"
#include "../include/parallelcore.cuh"

using namespace std;

void GNC(vector<byte *> &uData, vector<int> &uLens, vector<byte *> &uKeys, vector<byte *> &ciphers) {
    
    // The published algorithm copies the ciphers back to uData
    // But I'm gonna put them in a separate array in case I need the raw user data for something.

    // The following variables are stored in global memory
    // They will be further copied to shared memory in the kernel
    // The idea being to reduce memory latency 
    byte *d_sbox;
    byte *d_mul2;
    byte *d_mul3;

    gpuErrchk(hipMalloc((void **) &d_sbox, 256));
    gpuErrchk(hipMalloc((void **) &d_mul2, 256));
    gpuErrchk(hipMalloc((void **) &d_mul3, 256));

    gpuErrchk(hipMemcpy(d_sbox, sbox, 256, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_mul2, mul2, 256, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_mul3, mul3, 256, hipMemcpyHostToDevice));


    int n;
    byte expandedKey[176];
    byte *d_expandedKey;
    gpuErrchk(hipMalloc((void**) &d_expandedKey, 176));

    int gridsize, blocksize;
    for(int i = 0; i < uData.size(); i++) {
        n = uLens[i];
        byte *d_message;
        byte *cipher = new byte[n];
        gpuErrchk(hipMalloc((void**) &d_message, n));
        
        KeyExpansion(uKeys[i], expandedKey);
        gpuErrchk(hipMemcpy(d_expandedKey, expandedKey, 176, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_message, uData[i], n, hipMemcpyHostToDevice));
        
        blocksize = BLOCKSIZE;
        gridsize = ceil (uLens[i]/(BLOCKSIZE*16));
        
        if(uLens[i] <= BLOCKSIZE) gridsize = 1;

        GNC_Cipher <<< gridsize, blocksize>>> (d_message, n, d_expandedKey, d_sbox, d_mul2, d_mul3);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipMemcpy(cipher, d_message, n, hipMemcpyDeviceToHost));
        ciphers.push_back(move(cipher));

        gpuErrchk(hipFree(d_message));
    }
    
    // cout << endl << endl;
    // cout << hex(uData[0], uLens[0]);
    // cout << endl << endl << endl << endl;
    // cout << hex(ciphers[0], uLens[0]);
}

void get_data(opts vars, vector<byte*> &msgs, vector<int> &lens, vector<byte*> &keys, int i, int j) {

    if(i < vars.n_files_start || i > vars.n_files_end || j < 0 || j >= vars.m_batches ) {
        cout << "Invalid getdata params";
        return;
    }

	string msg_path, key_path;
    ifstream f_msg, f_key;

    int k, n;
    for(k = 0; k < i; k++) {
        msg_path = vars.path + "/" + to_string(i) + "/" + to_string(j) + "/" + to_string(k);
        key_path = msg_path+"_key";

        f_msg.open(msg_path, ios::binary);
        f_key.open(key_path, ios::binary);

	    if(f_msg && f_key) {

		    f_msg.seekg(0, f_msg.end);
	        n = f_msg.tellg();
    		f_msg.seekg(0, f_msg.beg);

            byte *message = new byte[n];
		    byte *key = new byte[16];

            f_msg.read( reinterpret_cast<char *> (message), n);
		    f_key.read( reinterpret_cast<char *> (key), 16);

            // if(k == 0) cout << endl << endl << hex(message, n) << endl << endl;

            msgs.push_back(move(message));
            lens.push_back(n);
            keys.push_back(move(key));

            f_msg.close();
            f_key.close();
        }
        else {
            cout << "read failed";
        }
    }

    // cout << msgs.size() << endl;
    // cout << hex(keys[i-1], 16) << endl;
    // cout << hex(msgs[0], lens[0]) << endl;
}


int main() {
    opts vars = get_defaults();
	clock_t start, end;
    int i, j;
    for(i = vars.n_files_start; i <= vars.n_files_end; i += vars.step) {
        
        long long isum = 0;
        for(j = 0; j < vars.m_batches; j++) {
            vector<long> batchtimes;
			long sum = 0;
            
            vector<byte*> uData;
            vector<int> uLens;
            vector<byte*> uKeys;

            get_data(vars, uData, uLens, uKeys, i, j);
            vector<byte*> ciphers;
            
            start = clock();
            GNC(uData, uLens, uKeys, ciphers);
            end = clock();
            batchtimes.push_back((end-start));
			sum += (end-start);
			printf("\n N_FILES: %5d | BATCH: %2d | TIME: %10.4lf ms", i, j, ((double)sum * 100)/CLOCKS_PER_SEC);
			isum += sum;

            string out_path;
            ofstream fout;
            for(int k = 0; k < i; k++) {
                out_path = vars.path + "/" + to_string(i) + "/" + to_string(j) + "/" + to_string(k) + "_cipher_gnc";
                fout.open(out_path, ios::binary);
                fout.write(reinterpret_cast<char *> (ciphers[k]), uLens[k]);
                fout.close();
                // free(uData[k]);
                // free(uKeys[k]);
                delete[] uData[k];
                delete[] uKeys[k];
            }
        }
		printf("\n N_FILES: %5d | AVG_TIME: %10.4lf ms\n", i, (((double)isum * 100)/vars.m_batches)/CLOCKS_PER_SEC);
    }

    return 0;
}

/*
    // VERIFICATION ANALYSIS
    byte *d_sbox;
    byte *d_mul2;
    byte *d_mul3;
    hipMalloc((void **) &d_sbox, 256);
    hipMalloc((void **) &d_mul2, 256);
    hipMalloc((void **) &d_mul3, 256);
    
    hipMemcpy(d_sbox, sbox, 256, hipMemcpyHostToDevice);
    hipMemcpy(d_mul2, mul2, 256, hipMemcpyHostToDevice);
    hipMemcpy(d_mul3, mul3, 256, hipMemcpyHostToDevice);
    
    byte message[] = {0x32, 0x43, 0xf6, 0xa8, 0x88, 0x5a, 0x30, 0x8d, 0x31, 0x31, 0x98, 0xa2, 0xe0, 0x37, 0x07, 0x34, 0x32, 0x43, 0xf6, 0xa8, 0x88, 0x5a, 0x30, 0x8d, 0x31, 0x31, 0x98, 0xa2, 0xe0, 0x37, 0x07, 0x34, 0x32, 0x43, 0xf6, 0xa8, 0x88, 0x5a, 0x30, 0x8d, 0x31, 0x31, 0x98, 0xa2, 0xe0, 0x37, 0x07, 0x34};
    byte key[] = {0x2b, 0x7e, 0x15, 0x16, 0x28, 0xae, 0xd2, 0xa6, 0xab, 0xf7, 0x15, 0x88, 0x09, 0xcf, 0x4f, 0x3c};
    byte expandedKey[176];
    byte cipher[48];
    byte* d_message;
    byte* d_expandedKey;
    // byte* d_cipher;
    int n = 48;
            
    // byte cipher[n];
    hipMalloc((void**) &d_message, n);
    // hipMalloc((void**) &d_cipher, n);
    hipMalloc((void**) &d_expandedKey, 176);
    
    KeyExpansion(key, expandedKey);
    hipMemcpy(d_expandedKey, expandedKey, 176, hipMemcpyHostToDevice);
    hipMemcpy(d_message, message, n, hipMemcpyHostToDevice);
    Cipher <<<1, 256>>> (d_message, n, d_expandedKey, d_sbox, d_mul2, d_mul3);

    hipMemcpy(cipher, d_message, n, hipMemcpyDeviceToHost);
    cout << hex(cipher, 48) << endl;  
*/