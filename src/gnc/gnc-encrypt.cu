#include <iostream>
#include <string>
#include <stdlib.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include <vector>

#include "../include/aeslib.h"
#include "../include/genlib.hpp"
#include "../include/parallelcore.cuh"

using namespace std;

void get_data(opts vars, vector<byte*> &msgs, vector<int> &lens, vector<byte*> &keys, int i, int j) {

    if(i < vars.n_files_start || i > vars.n_files_end || j < 0 || j >= vars.m_batches ) {
        cout << "Invalid getdata params";
        return;
    }

	string msg_path, key_path;
    ifstream f_msg, f_key;

    for(k = 0; k < i; k++) {
        msg_path = vars.path + "/" + to_string(i) + "/" + to_string(j) + "/" + to_string(k);
        key_path = path+"_key";

        cout << msg_path << " ";
        f_msg.open(msg_path);
        f_key.open(key_path);

	    if(f_msg && f_key) {

		    f_msg.seekg(0, f_msg.end);
	        n = f_msg.tellg();
            cout << n << endl;
    		f_msg.seekg(0, f_msg.beg);

            byte message[n];
		    byte key[16];

            f_msg.read(reinterpret_cast<char *> (message), n);
		    f_key.read(reinterpret_cast<char *> (key), 16);

            msgs.push_back(message);
            lens.push_back(n);
            keys.push_back(key);

            f_msg.close();
            f_key.close();
        }
        else {
            cout << "read failed";
        }
    }
}

void GNC(vector<byte *> &uData, vector<int> &uLens vector<byte *> &uKeys, vector<byte *> &ciphers) {
    
    // The published algorithm copies the ciphers back to uData
    // But I'm gonna put them in a separate array in case I need the raw user data for something.

    
    // The following variables are stored in global memory
    // They will be further copied to shared memory in the kernel
    // The idea being to reduce memory latency 
    byte* d_sbox;
    byte* d_mul2;
    byte* d_mul3;
    load_boxes(d_sbox, d_mul2, d_mul3);

    int n;
    byte expandedKey[176];
    byte* d_expandedKey;
    hipMalloc((void**) &d_expandedKey, 176);

    byte *message;
    byte *cipher;

    for(int i = 0; i < uData.length(); i++) {
        n = uLens[i];
        byte message[n];
        byte cipher[n];
        


        KeyExpansion(key, expandedKey);
        hipMemcpy(d_expandedKey, expandedKey, 176, hipMemcpyHostToDevice);

	    Cipher(message, n, expandedKey, cipher);
    }
	// cout << "MSG\n" << hex(message, n) << endl << endl; 
	// cout << "Cry\n" << hex(cipher, n) << endl << endl;	
}

